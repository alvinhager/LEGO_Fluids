#include "hip/hip_runtime.h"
#include "CUDAFunctions.h"

__device__ int  getFlatIndex(int i, int j, int k, int width, int height)
{
        return (unsigned int)i + (unsigned int)width * ((unsigned int)j + (unsigned int)height * (unsigned int)k);
};

__global__ void kernel6(float* velocity_field, 
                        float* weights, 
                        float* dx, FluidParticle* particles,
                        int *no_particles,
                        int* direction,
                        int* v_width, int* v_height, int* v_depth
                        )
{

  int U = 0;
  int V = 1;
  int W = 2;

    VectorMath::vec3 offset;
    float hdx = (float)(0.5 * (*dx));
    if (*direction == U)
        offset = VectorMath::vec3(0.0f, hdx, hdx);
    else if (*direction == V)
        offset = VectorMath::vec3(hdx, 0.0f, hdx);
    else if (*direction == W)
        offset = VectorMath::vec3(hdx, hdx, 0.0f);
    else
        return;

    float r = *dx;
    float rsq = r * r;
    float coef1 = (4.0f / 9.0f) * (1.0f / (r * r * r * r * r * r));
    float coef2 = (17.0f / 9.0f) * (1.0f / (r * r * r * r));
    float coef3 = (22.0f / 9.0f) * (1.0f / (r * r));

    // transfer particle velocity component to grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int n=*no_particles;
    
    for (int pidx = index; pidx < n; pidx+=stride)
    {
     
        VectorMath::vec3 p = particles[pidx].position - offset;
        float velocityComponent = particles[pidx].velocity[*direction];

        GridIndex g = Grid3D::positionToGridIndex(p, *dx);
        GridIndex gmin((int)fmax(g.i - 1.0, 0.0), (int)fmax(g.j - 1.0, 0.0), (int)fmax(g.k - 1.0, 0.0));
        GridIndex gmax((int)fmin(g.i + 1.0, *v_width - 1.0),(int)fmin(g.j + 1.0, *v_height - 1.0), (int)fmin(g.k + 1.0, *v_depth - 1.0));

        for (int k = gmin.k; k <= gmax.k; k++)
        {
            for (int j = gmin.j; j <= gmax.j; j++)
            {
                for (int i = gmin.i; i <= gmax.i; i++)
                {
                    VectorMath::vec3 gpos = Grid3D::GridIndexToPosition(i, j, k, *dx);
                    VectorMath::vec3 v = gpos - p;
                    float distsq = v.x * v.x + v.y * v.y + v.z * v.z;
                    if (distsq < rsq)
                    {
                        float weight = 1.0f - coef1 * distsq * distsq * distsq + coef2 * distsq * distsq - coef3 * distsq;
                        int idx=getFlatIndex(i,j,k, *v_width, *v_height);
                        atomicAdd(&velocity_field[idx], weight*velocityComponent);
                        atomicAdd(&weights[idx], weight);
                    }
                }
            }
         }

      
     }

};


void computeVelocityFieldAndWeightsWithGPU(Array3D<float> &velocity_field,Array3D<bool> &isValueSet,int direction,FluidSimulation &fluidsim, Array3D<float> &weights )
{
 FluidParticle* h_particles= &fluidsim.particles[0];
 float *d_velocity_field, *d_weights, 
 
 float * d_dx;
 float dx=fluidsim.getdx();
 float* h_dx=&dx;

 int no_particles = fluidsim.particles.size();

 FluidParticle* d_particles;

 int *d_direction;
 bool *d_isValueSet;
 int* d_v_width, *d_v_height,*d_v_depth, 
 int* d_no_particles;

  hipMalloc((void**)&d_velocity_field, sizeof(float)*velocity_field.getSize());
  hipMemcpy(d_velocity_field, velocity_field.getRawArray(),sizeof(float)*velocity_field.getSize(), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_weights, sizeof(float)*weights.getSize());
  hipMemcpy(d_weights, weights.getRawArray(),sizeof(float)*weights.getSize(), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dx, sizeof(float));
  hipMemcpy(d_dx, &dx, sizeof(float), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_particles, sizeof(FluidParticle)*fluidsim.particles.size());
  hipMemcpy(d_particles, h_particles, sizeof(FluidParticle)*fluidsim.particles.size(), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_direction, sizeof(int));
  hipMemcpy(d_direction, &direction, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_no_particles, sizeof(int));
  hipMemcpy(d_no_particles,&no_particles, sizeof(int), hipMemcpyHostToDevice);

  // dimensions of grid passed to GPU
  hipMalloc((void**)&d_v_width, sizeof(int));
  hipMemcpy(d_v_width, &velocity_field.width, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_v_height, sizeof(int));
  hipMemcpy(d_v_height, &velocity_field.height, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_v_depth, sizeof(int));
  hipMemcpy(d_v_depth, &velocity_field.depth, sizeof(int), hipMemcpyHostToDevice);

int N=fluidsim.particles.size();
int blockSize=256; 
int numBlocks = (N+ blockSize - 1)/blockSize;

kernel6<<<blockSize,numBlocks>>>(d_velocity_field, d_weights, d_dx, d_particles, d_no_particles, d_direction, d_v_width,d_v_height, d_v_depth);

hipDeviceSynchronize();

hipMemcpy(velocity_field.getRawArray(), d_velocity_field, sizeof(float) * velocity_field.getSize(), hipMemcpyDeviceToHost);
hipMemcpy(weights.getRawArray(), d_weights, sizeof(float) * weights.getSize(), hipMemcpyDeviceToHost);

// free device memory
hipFree(d_velocity_field);
hipFree(d_weights);
hipFree(d_weights);
hipFree(d_dx);
hipFree(d_v_width);
hipFree(d_v_height);
hipFree(d_v_depth);
hipFree(d_particles);
hipFree(d_no_particles);
};
